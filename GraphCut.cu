#include "hip/hip_runtime.h"
/*
 * GraphCut.cu
 *
 *  Created on: Jun 8, 2012
 *      Author: bruno
 */


#ifndef GRAPHCUT_CU_
#define GRAPHCUT_CU_

#include <cstdlib>
#include <cmath>
#include <cstring>
#include <cassert>

#include "GraphCut.h"
#include "GraphCutKernels.cu"

#ifdef DEBUG_MODE

NodeWrapper host_copy;

static void initialize_graph(GraphWrapper gw) {
	host_copy.height = (int *) malloc(sizeof(int)*gw.width_ex*gw.height_ex);
	host_copy.excess = (int *) malloc(sizeof(int)*gw.width_ex*gw.height_ex);
	host_copy.status = (int *) malloc(sizeof(int)*gw.width_ex*gw.height_ex);
	host_copy.comp_h = (int *) malloc(sizeof(int)*gw.width_ex*gw.height_ex);
	host_copy.comp_n = (int *) malloc(sizeof(int)*gw.width_ex*gw.height_ex);
	host_copy.edge_l = (int *) malloc(sizeof(int)*gw.width_ex*gw.height_ex);
	host_copy.edge_r = (int *) malloc(sizeof(int)*gw.width_ex*gw.height_ex);
	host_copy.edge_u = (int *) malloc(sizeof(int)*gw.width_ex*gw.height_ex);
	host_copy.edge_d = (int *) malloc(sizeof(int)*gw.width_ex*gw.height_ex);
#if NEIGHBORHOOD == 8
	host_copy.edge_ul = (int *) malloc(sizeof(int)*gw.width_ex*gw.height_ex);
	host_copy.edge_dr = (int *) malloc(sizeof(int)*gw.width_ex*gw.height_ex);
	host_copy.edge_ur = (int *) malloc(sizeof(int)*gw.width_ex*gw.height_ex);
	host_copy.edge_dl = (int *) malloc(sizeof(int)*gw.width_ex*gw.height_ex);
#endif
}

static void update_graph(GraphWrapper gw) {
	CUDA_SAFE_CALL(hipMemcpy(host_copy.height,gw.n.height,sizeof(int)*gw.width_ex*gw.height_ex,hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(host_copy.excess,gw.n.excess,sizeof(int)*gw.width_ex*gw.height_ex,hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(host_copy.status,gw.n.status,sizeof(int)*gw.width_ex*gw.height_ex,hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(host_copy.comp_h,gw.n.comp_h,sizeof(int)*gw.width_ex*gw.height_ex,hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(host_copy.comp_n,gw.n.comp_n,sizeof(int)*gw.width_ex*gw.height_ex,hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(host_copy.edge_l,gw.n.edge_l,sizeof(int)*gw.width_ex*gw.height_ex,hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(host_copy.edge_r,gw.n.edge_r,sizeof(int)*gw.width_ex*gw.height_ex,hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(host_copy.edge_u,gw.n.edge_u,sizeof(int)*gw.width_ex*gw.height_ex,hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(host_copy.edge_d,gw.n.edge_d,sizeof(int)*gw.width_ex*gw.height_ex,hipMemcpyDeviceToHost));
#if NEIGHBORHOOD == 8
	CUDA_SAFE_CALL(hipMemcpy(host_copy.edge_ul,gw.n.edge_ul,sizeof(int)*gw.width_ex*gw.height_ex,hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(host_copy.edge_dr,gw.n.edge_dr,sizeof(int)*gw.width_ex*gw.height_ex,hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(host_copy.edge_ur,gw.n.edge_ur,sizeof(int)*gw.width_ex*gw.height_ex,hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(host_copy.edge_dl,gw.n.edge_dl,sizeof(int)*gw.width_ex*gw.height_ex,hipMemcpyDeviceToHost));
#endif
}

static void print_graph(GraphWrapper gw) {
	update_graph(gw);
	for(int i = 0 ; i < gw.height ; ++i) {
		for(int j = 0 ; j < gw.width ; ++j) {
			printf("%2d ",host_copy.height[i*gw.width_ex + j]);
		}
		printf(" | ");
		for(int j = 0 ; j < gw.width ; ++j) {
			printf("%2d ",host_copy.excess[i*gw.width_ex + j]);
		}
		printf(" | ");
		for(int j = 0 ; j < gw.width ; ++j) {
			printf("%2d ",host_copy.edge_u[i*gw.width_ex + j]);
		}
		printf(" | ");
		for(int j = 0 ; j < gw.width ; ++j) {
			printf("%2d ",host_copy.edge_l[i*gw.width_ex + j]);
		}
		printf(" | ");
		for(int j = 0 ; j < gw.width ; ++j) {
			printf("%2d ",host_copy.edge_d[i*gw.width_ex + j]);
		}
		printf(" | ");
		for(int j = 0 ; j < gw.width ; ++j) {
			printf("%2d ",host_copy.edge_r[i*gw.width_ex + j]);
		}
		printf(" | ");
		for(int j = 0 ; j < gw.width ; ++j) {
			printf("%2d ",host_copy.comp_h[i*gw.width_ex + j]);
		}
		printf(" | ");
		for(int j = 0 ; j < gw.width ; ++j) {
			printf("%2d ",host_copy.comp_n[i*gw.width_ex + j]);
		}
		printf("\n");
	}
}

static void free_graph(GraphWrapper gw) {
	free(host_copy.height);
	free(host_copy.excess);
	free(host_copy.status);
	free(host_copy.comp_h);
	free(host_copy.comp_n);
	free(host_copy.edge_l);
	free(host_copy.edge_r);
	free(host_copy.edge_u);
	free(host_copy.edge_d);
#if NEIGHBORHOOD == 8
	free(host_copy.edge_ul);
	free(host_copy.edge_dr);
	free(host_copy.edge_ur);
	free(host_copy.edge_dl);
#endif
}

#endif

#define ROUND_UP(a,b) ((int)ceil((float)a/(float)b))
#define MAKE_DIVISIBLE(a,b) (b*ROUND_UP(a,b))

GlobalWrapper GC_Init(int width, int height, bool full_arguments = false, int * data_positive = NULL, int * data_negative = NULL, int penalty = 0)
{
	GlobalWrapper ret;
	KernelWrapper ker;

	assert(THREADS_X == 32 && THREADS_Y == 8);

	ker.g.width = width;
	ker.g.height = height;
	ker.g.size = width*height;
	ker.g.width_ex = MAKE_DIVISIBLE(width,THREADS_X);
	ker.g.height_ex = MAKE_DIVISIBLE(height,THREADS_Y);
	ker.g.size_ex = ker.g.width_ex * ker.g.height_ex;

	ker.block_x = ROUND_UP(ker.g.width_ex,THREADS_X);
	ret.block_y = ROUND_UP(ker.g.height_ex,THREADS_Y);
	ret.block_count = ROUND_UP(ker.g.size_ex,THREAD_COUNT);

	ret.penalty = penalty;

	CUDA_SAFE_CALL(hipMalloc((void**)&(ker.g.n.edge_u),sizeof(int)*ker.g.size_ex));
	CUDA_SAFE_CALL(hipMalloc((void**)&(ker.g.n.edge_d),sizeof(int)*ker.g.size_ex));
	CUDA_SAFE_CALL(hipMalloc((void**)&(ker.g.n.edge_l),sizeof(int)*ker.g.size_ex));
	CUDA_SAFE_CALL(hipMalloc((void**)&(ker.g.n.edge_r),sizeof(int)*ker.g.size_ex));
#if NEIGHBORHOOD == 8
	CUDA_SAFE_CALL(hipMalloc((void**)&(ker.g.n.edge_ul),sizeof(int)*ker.g.size_ex));
	CUDA_SAFE_CALL(hipMalloc((void**)&(ker.g.n.edge_ur),sizeof(int)*ker.g.size_ex));
	CUDA_SAFE_CALL(hipMalloc((void**)&(ker.g.n.edge_dl),sizeof(int)*ker.g.size_ex));
	CUDA_SAFE_CALL(hipMalloc((void**)&(ker.g.n.edge_dr),sizeof(int)*ker.g.size_ex));
#endif
	CUDA_SAFE_CALL(hipMalloc((void**)&(ker.g.n.height),sizeof(int)*ker.g.size_ex));
	CUDA_SAFE_CALL(hipMalloc((void**)&(ker.g.n.excess),sizeof(int)*ker.g.size_ex));
	CUDA_SAFE_CALL(hipMalloc((void**)&(ker.g.n.status),sizeof(int)*ker.g.size_ex));
	CUDA_SAFE_CALL(hipMalloc((void**)&(ker.g.n.comp_h),sizeof(int)*ker.g.size_ex));
	CUDA_SAFE_CALL(hipMalloc((void**)&(ker.g.n.comp_n),sizeof(int)*ker.g.size_ex));
	CUDA_SAFE_CALL(hipMalloc((void**)&(ker.active),sizeof(int)*ret.block_count));

	CUDA_SAFE_CALL(hipMalloc((void**)&(ret.data_positive),sizeof(int)*width*height));
	CUDA_SAFE_CALL(hipMalloc((void**)&(ret.data_negative),sizeof(int)*width*height));

	if (full_arguments)
	{
		CUDA_SAFE_CALL(hipMemcpy(ret.data_positive,data_positive,sizeof(int)*width*height,hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(ret.data_negative,data_negative,sizeof(int)*width*height,hipMemcpyHostToDevice));
		GC_SetGraph(ret);
	}

	ret.k = ker;

	return ret;
}

void GC_SetDataterms(GlobalWrapper* gw, int* data_positive, int* data_negative)
{
	//Term::getNeg();
	gw->data_positive = data_positive;
	gw->data_negative = data_negative;
	/*gw->data_positive = DataTerm::getPos();
	gw->data_negative = DataTerm::getNeg();*/

	//CUDA_SAFE_CALL(hipMemcpy(gw.data_positive,data_positive,sizeof(int)*gw.k.g.size,hipMemcpyHostToDevice));
	//CUDA_SAFE_CALL(hipMemcpy(gw.data_negative,data_negative,sizeof(int)*gw.k.g.size,hipMemcpyHostToDevice));
}

void GC_SetPenalty(GlobalWrapper* gw, int p)
{
	gw->penalty = p;
}

void GC_SetGraph(GlobalWrapper gw)
{

	dim3 block(THREADS_X,THREADS_Y,1);
	dim3 grid(gw.k.block_x, gw.block_y,1);

	printf("pen %d\n", gw.penalty);
	InitGraph<<<grid,block>>>(gw.k, gw.data_positive, gw.data_negative, gw.penalty);
	cutilCheckMsg("InitGraph kernel launch failure");
}

void GC_Update(GlobalWrapper gw, int * data) {
	assert(gw.k.block_x);

}

#define ACTIVITY_CHECK_FREQUENCY 10
#define GLOBAL_RELABEL_FREQUENCY 1500 // 150
#define FIRST_GLOBAL_RELABEL 200
#define PUSHES_PER_RELABEL 8

void GC_Optimize(GlobalWrapper gw, int * label) {
	dim3 block(THREADS_X,THREADS_Y,1);
	dim3 grid(gw.k.block_x,gw.block_y,1);
	
#ifdef DEBUG_MODE
	initialize_graph(gw.k.g);
#endif

	int * zero_arr = (int *) malloc(8*sizeof(int));
	zero_arr[0] = 0;
	int * h_alive = (int *) malloc(8*sizeof(int));
	int * d_alive;
	CUDA_SAFE_CALL(hipMalloc((void**)&(d_alive),8*sizeof(int)));
	
	int counter = 0;
	
	//const char * error;
	
	int * d_heights;
	CUDA_SAFE_CALL(hipMalloc((void**)&(d_heights),(gw.k.g.size_ex+10)*sizeof(int)));

	//bool dbg_verify_no_more_pushes = false;
	
#ifdef DEBUG_MODE
		CUDA_SAFE_CALL(hipDeviceSynchronize());
		printf("Starting out:\n");
		print_graph(gw.k.g);
#endif

	unsigned int timer = 0;
	CUT_SAFE_CALL(cutCreateTimer(&timer));
	CUT_SAFE_CALL(cutStartTimer(timer));

	while(1) {
		int skip = counter % ACTIVITY_CHECK_FREQUENCY;
		
		++counter;
		
		if(!skip)
			CUDA_SAFE_CALL(hipMemcpy(d_alive,zero_arr,8*sizeof(int),hipMemcpyHostToDevice));
		
		Relabel<<<grid,block>>>(gw.k, skip);
		/*cutilCheckMsg("Relabel kernel launch failure");error = hipGetErrorString(hipPeekAtLastError());
		printf("%s\nwith %d iters.\n", error,counter);
		error = hipGetErrorString(hipDeviceSynchronize());
		printf("%s\n", error);*/
		
		//CUDA_SAFE_CALL(hipDeviceSynchronize());
		
#ifdef DEBUG_MODE
		CUDA_SAFE_CALL(hipDeviceSynchronize());
		printf("After Relabel:\n");
		print_graph(gw.k.g);
#endif

		Push<<<grid,block>>>(gw.k, PUSHES_PER_RELABEL, skip, d_alive);
		/*cutilCheckMsg("Push kernel launch failure");error = hipGetErrorString(hipPeekAtLastError());
		printf("%s\nwith %d iters.\n", error,counter);
		error = hipGetErrorString(hipDeviceSynchronize());
		printf("%s\n", error);*/
		
		//CUDA_SAFE_CALL(hipDeviceSynchronize());
		
#ifdef DEBUG_MODE
		CUDA_SAFE_CALL(hipDeviceSynchronize());
		printf("After Push:\n");
		print_graph(gw.k.g);
#endif

		if(!skip)
			CUDA_SAFE_CALL(hipMemcpy(h_alive,d_alive,8*sizeof(int),hipMemcpyDeviceToHost));
	
	
		if(!skip) {
			UpdateActivity<<<grid,block>>>(gw.k.g.n.status, gw.k.active, gw.k.block_x, gw.k.g.width_ex);
			/*cutilCheckMsg("UpdateActivity kernel launch failure");error = hipGetErrorString(hipPeekAtLastError());
			printf("%s\nwith %d iters.\n", error,counter);
			error = hipGetErrorString(hipDeviceSynchronize());
			printf("%s\n", error);*/
		}
		//CUDA_SAFE_CALL(hipDeviceSynchronize());
		
		if(!h_alive[0]) {
			//dbg_verify_no_more_pushes = true;
			break;
		} /*else if ( dbg_verify_no_more_pushes ) {
			printf("Became alive after being dead!!\n");
			assert(false);
		}*/
		if(!((counter - FIRST_GLOBAL_RELABEL) % GLOBAL_RELABEL_FREQUENCY)) {
			InitGlobalRelabel<<<grid,block>>>(gw.k);
			cutilCheckMsg("InitGlobalRelabel kernel launch failure");
			int iter_gr = 0;
			while(1){
				//if(!(iter_gr % GLOBAL_RELABEL_CHECK_FREQUENCY))
					CUDA_SAFE_CALL(hipMemcpy(d_alive,zero_arr,8*sizeof(int),hipMemcpyHostToDevice));
				GlobalRelabel<<<grid,block>>>(gw.k, d_alive);
				cutilCheckMsg("GlobalRelabel kernel launch failure");
				//if(!(iter_gr % GLOBAL_RELABEL_CHECK_FREQUENCY))
					CUDA_SAFE_CALL(hipMemcpy(h_alive,d_alive,8*sizeof(int),hipMemcpyDeviceToHost));
				if(!h_alive[0])
					break;
				iter_gr++;
			}
			printf("%d iterations inside global relabel\n",iter_gr);
			h_alive[0] = 1;
#ifdef DEBUG_MODE
			CUDA_SAFE_CALL(hipDeviceSynchronize());
			printf("After Global Relabel:\n");
			print_graph(gw.k.g);
#endif
		}

		/*if(counter > 20000) {
			printf("Too long inside the main loop\n");
			break;
		}*/
		if(!(counter % 500))
			printf("counter: %d\n",counter);
	}
	
#ifdef DEBUG_MODE
	free_graph(gw.k.g);
#endif

	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUT_SAFE_CALL(cutStopTimer(timer));
	printf("Graph Cut used %d iterations and %f milliseconds.\n", counter, cutGetTimerValue(timer));
	CUT_SAFE_CALL(cutDeleteTimer(timer));
	
	CUDA_SAFE_CALL(hipFree(d_heights));

	int * d_label;
	CUDA_SAFE_CALL(hipMalloc((void**)&(d_label),sizeof(int)*gw.k.g.width*gw.k.g.height));
	CUDA_SAFE_CALL(hipMemcpy(d_label,label,sizeof(int)*gw.k.g.width*gw.k.g.height,hipMemcpyHostToDevice));
	InitLabels<<<grid,block>>>(gw.k, d_label);
	cutilCheckMsg("InitLabels kernel launch failure");
	
	int spreadLabelsCounter = 0;
	while(1){
		spreadLabelsCounter++;
		CUDA_SAFE_CALL(hipMemcpy(d_alive,zero_arr,8*sizeof(int),hipMemcpyHostToDevice));
		SpreadLabels<<<grid,block>>>(gw.k, d_label, d_alive);
		cutilCheckMsg("SpreadLabels kernel launch failure");
		CUDA_SAFE_CALL(hipMemcpy(h_alive,d_alive,8*sizeof(int),hipMemcpyDeviceToHost));
		if(!h_alive[0])
			break;
	}
	printf("%d iterations inside Spreadlabels\n",spreadLabelsCounter);
	CUDA_SAFE_CALL(hipMemcpy(label,d_label,sizeof(int)*gw.k.g.width*gw.k.g.height,hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_label));
	CUDA_SAFE_CALL(hipFree(d_alive));
	free(h_alive);
	free(zero_arr);
}

void GC_End(GlobalWrapper * gw) {
	GlobalWrapper clean;

	CUDA_SAFE_CALL(hipFree(gw->k.g.n.edge_u));
	CUDA_SAFE_CALL(hipFree(gw->k.g.n.edge_d));
	CUDA_SAFE_CALL(hipFree(gw->k.g.n.edge_l));
	CUDA_SAFE_CALL(hipFree(gw->k.g.n.edge_r));
#if NEIGHBORHOOD == 8
	CUDA_SAFE_CALL(hipFree(gw->k.g.n.edge_ul));
	CUDA_SAFE_CALL(hipFree(gw->k.g.n.edge_ur));
	CUDA_SAFE_CALL(hipFree(gw->k.g.n.edge_dl));
	CUDA_SAFE_CALL(hipFree(gw->k.g.n.edge_dr));
#endif
	CUDA_SAFE_CALL(hipFree(gw->k.g.n.height));
	CUDA_SAFE_CALL(hipFree(gw->k.g.n.excess));
	CUDA_SAFE_CALL(hipFree(gw->k.g.n.status));
	CUDA_SAFE_CALL(hipFree(gw->k.g.n.comp_h));
	CUDA_SAFE_CALL(hipFree(gw->k.g.n.comp_n));
	CUDA_SAFE_CALL(hipFree(gw->k.active));
	
	CUDA_SAFE_CALL(hipFree(gw->data_positive));
	CUDA_SAFE_CALL(hipFree(gw->data_negative));

	clean.k.block_x = clean.block_y = clean.k.g.width = clean.k.g.height = 0;
	*gw = clean;
}


#endif /* GRAPHCUT_CU_ */
