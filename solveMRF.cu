#include <iostream>
#include <ctime>
#include <fstream>
#include <cstdio>
#include <cstdlib>
#include <cassert>

using namespace std;

void loadMiddleburyMRFData(const char * filename, int* &data_positive, int* &data_negative, int* &hCue, int* &vCue, int &width, int &height, int &nLabels)
{
	FILE * fp;
	fp = fopen(filename,"rb");
	assert(fp);

	assert(fscanf(fp,"%d %d %d",&width,&height,&nLabels)==3);

	int i, gt;
	for(i = 0; i < width * height; i++)
		assert(fscanf(fp,"%d",&gt)==1);
	assert(gt == gt);

	assert(nLabels == 2);

	data_positive = (int*) malloc(width * height * sizeof(int));
	data_negative = (int*) malloc(width * height * sizeof(int));
	assert(data_positive && data_negative);

	int v;

	for(i = 0; i < width * height; i++) {
		assert(fscanf(fp,"%d",&v)==1);
		data_positive[i] = v;
	}

	for(i = 0; i < width * height; i++) {
		assert(fscanf(fp,"%d",&v)==1);
		data_negative[i] = v;
	}

	hCue = (int*) malloc(width * height * sizeof(int));
	vCue = (int*) malloc(width * height * sizeof(int));
	assert(hCue && vCue);

	int x, y;
	for(y = 0; y < height; y++) {
		for(x = 0; x < width-1; x++) {
			assert(fscanf(fp,"%d",&v)==1);
			hCue[x+y*width] = v;
		}
	}

	for(y = 0; y < height-1; y++) {
		for(x = 0; x < width; x++) {
			assert(fscanf(fp,"%d",&v)==1);
			vCue[y*width+x] = v;
		}
	}
	for(x = 0; x < width; x++) {
		vCue[(height-1)*width+x] = 0;
		hCue[(height-1)*width+x] = 0;
	}

	fclose(fp);
}

#include "GraphCut.cu"

int main(int argc, char * argv[]) {
	if(argc != 2) {
		printf("Usage: %s MDF_file\n",argv[0]);
		exit(1);
	}
	int* data_positive, * data_negative, * hCue, * vCue, width, height, nLabels;
	loadMiddleburyMRFData(argv[1],data_positive,data_negative,hCue,vCue,width,height,nLabels);

	int * d_data_positive, * d_data_negative, * d_up, * d_down, * d_left, * d_right;

	CUDA_SAFE_CALL(hipMalloc((void**)&(d_data_positive),sizeof(int)*width*height));
	CUDA_SAFE_CALL(hipMalloc((void**)&(d_data_negative),sizeof(int)*width*height));
	CUDA_SAFE_CALL(hipMalloc((void**)&(d_up),sizeof(int)*width*height));
	CUDA_SAFE_CALL(hipMalloc((void**)&(d_down),sizeof(int)*width*height));
	CUDA_SAFE_CALL(hipMalloc((void**)&(d_left),sizeof(int)*width*height));
	CUDA_SAFE_CALL(hipMalloc((void**)&(d_right),sizeof(int)*width*height));
	CUDA_SAFE_CALL(hipMemcpy(d_data_positive,data_positive,sizeof(int)*width*height,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_data_negative,data_negative,sizeof(int)*width*height,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_up,vCue,sizeof(int)*width*height,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_down,vCue,sizeof(int)*width*height,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_left,hCue,sizeof(int)*width*height,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_right,hCue,sizeof(int)*width*height,hipMemcpyHostToDevice));

	free(data_positive);
	free(data_negative);
	free(hCue);
	free(vCue);

	srand( time(NULL));

	printf("Solving a %d x %d MRF problem...\n",height,width);

	if(NEIGHBORHOOD != 4)
		printf("Please change NEIGHBORHOOD to 4\n");
	assert(NEIGHBORHOOD == 4);

	GlobalWrapper gw =
			GC_Init(width, height, d_data_positive, d_data_negative, 0, d_up, d_down, d_left, d_right);

	int * label = (int *) malloc(sizeof(int) * width * height);
	assert(label);

	GC_Optimize(gw, label);

	// print processed image
	ofstream face_out;
	face_out.open("labelMRF.ppm");
	face_out << "P3 " << width << " " << height << " 255 " << endl;
	for (unsigned i = 0; i < height; i++) {
		for (unsigned j = 0; j < width; j++) {
			if (label[i * width + j]) {
				face_out << 255 << " " << 255 << " " << 255 << " ";
			} else {
				face_out << 0 << " " << 0 << " " << 0 << " ";
			}
		}
		face_out << endl;
	}
	face_out.close();

	free(label);

	CUDA_SAFE_CALL(hipFree(d_data_positive));
	CUDA_SAFE_CALL(hipFree(d_data_negative));
	CUDA_SAFE_CALL(hipFree(d_up));
	CUDA_SAFE_CALL(hipFree(d_down));
	CUDA_SAFE_CALL(hipFree(d_left));
	CUDA_SAFE_CALL(hipFree(d_right));

	GC_End(&gw);

	return 0;
}
