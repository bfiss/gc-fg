#include "hip/hip_runtime.h"
/*
 * GraphCutKernels.cu
 *
 *  Created on: Jun 8, 2012
 *      Author: bruno
 */

#ifndef GRAPHCUTKERNELS_CU_
#define GRAPHCUTKERNELS_CU_

#include <stdio.h>

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
//#define printf(f, ...) ((void)(f, __VA_ARGS__),0)
#endif

#define DO_PUSH_C(edge,edge_inv,x_min,x_max,y_min,y_max,x_gap,y_gap,comp_h_idx)                                          \
	do{                                                                                              \
		excess = local_excess[local_idx];     \
		cap = edge[thread_id];                                                                       \
		bool changed = false; \
		if(cap > 0 && excess > 0 && /*local_height[local_idx] == */                                      \
           /*local_height[local_idx + (x_gap) + (y_gap) * 34] + 1*/ (comp_h&(1<<(comp_h_idx)))  ) {                                   \
			flow = excess > cap ? cap : excess;                                                      \
			/*excess -= flow;                                                                          \*/ \
			edge[thread_id] -= flow;                                                                 \
			edge_inv[thread_id + (x_gap) + (y_gap) * k.g.width_ex] += flow;                              \
			/*atomicSub(&k.g.n.excess[thread_id], flow);                                               \*/ \
			/*atomicAdd(&k.g.n.excess[thread_id + (x_gap) + (y_gap) * k.g.width_ex], flow);            \*/ \
			local_excess[local_idx] -= flow; \
			did_something = changed = true;                                                                    \
		}  \
		__syncthreads(); \
		changed ? local_excess[local_idx + (x_gap) + (y_gap) * 34] += flow : 0; \
		__syncthreads(); \
	} while(0)

#define DO_PUSH(edge,edge_inv,x_min,x_max,y_min,y_max,comp_h_idx) DO_PUSH_C(edge,edge_inv,x_min,x_max,y_min,y_max,(x_max-x_min),(y_max-y_min),comp_h_idx)

__global__ void InitGraph(KernelWrapper k, int * data_positive, int * data_negative, int penalty) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int thread_id = x + y * k.g.width_ex;
	int block_id = blockIdx.x + blockIdx.y * k.block_x;
	k.active[block_id] = 0;

	__syncthreads();

	bool inside_area = x > 0 && y > 0 && x < k.g.width - 1 && y < k.g.height - 1;

	inside_area ? k.active[block_id] = 1 : 0;
	int coming = inside_area ? data_positive[x + y * k.g.width] : 0;
	int going = inside_area ? data_negative[x + y * k.g.width] : 0;

	k.g.n.edge_u[thread_id] = inside_area ? penalty : 0;
	k.g.n.edge_d[thread_id] = inside_area ? penalty : 0;
	k.g.n.edge_l[thread_id] = inside_area ? penalty : 0;
	k.g.n.edge_r[thread_id] = inside_area ? penalty : 0;
#if NEIGHBORHOOD == 8
	k.g.n.edge_ul[thread_id] = inside_area ? penalty : 0;
	k.g.n.edge_ur[thread_id] = inside_area ? penalty : 0;
	k.g.n.edge_dl[thread_id] = inside_area ? penalty : 0;
	k.g.n.edge_dr[thread_id] = inside_area ? penalty : 0;
#endif
	k.g.n.height[thread_id] = going >= coming ? 1 : 2;
	k.g.n.excess[thread_id] = coming - going;
	k.g.n.status[thread_id] = going >= coming ? 0 : 1;
	
	x == 1             ? k.g.n.edge_l[thread_id] = 0 : 0;
	x == k.g.width - 2 ? k.g.n.edge_r[thread_id] = 0 : 0;
	y == 1             ? k.g.n.edge_u[thread_id] = 0 : 0;
	y == k.g.height - 2 ? k.g.n.edge_d[thread_id] = 0 : 0;
#if NEIGHBORHOOD == 8
	x == 1             ? k.g.n.edge_ul[thread_id] = k.g.n.edge_dl[thread_id] = 0 : 0;
	x == k.g.width - 2 ? k.g.n.edge_ur[thread_id] = k.g.n.edge_dr[thread_id] = 0 : 0;
	y == 1             ? k.g.n.edge_ul[thread_id] = k.g.n.edge_ur[thread_id] = 0 : 0;
	y == k.g.height - 2 ? k.g.n.edge_dr[thread_id] = k.g.n.edge_dl[thread_id] = 0 : 0;
#endif
}

#ifdef SPREAD_ZEROS
#define UPDATE_COMP_N_LABEL(i,edge) comp_n |= (1<<(i)) * (!edge[thread_id])
#endif

__global__ void InitLabels(KernelWrapper k, int * label) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int thread_id = x + y * k.g.width_ex;
	int label_id = x + y * k.g.width;
	
	if (x > 0 && y > 0 && x < k.g.width - 1 && y < k.g.height - 1) {
		//label[label_id] = k.g.n.height[thread_id] > k.g.size;
		//label[label_id] = k.g.n.excess[thread_id] > 0;
#ifdef SPREAD_ZEROS
		label[label_id] = k.g.n.excess[thread_id] >= 0;
#else
		label[label_id] = k.g.n.excess[thread_id] > 0;
#endif
	} else if(x < k.g.width && y < k.g.height) {
		label[label_id] = 0;
	}

	int comp_n = 0;
	UPDATE_COMP_N_LABEL(0,k.g.n.edge_d);
	UPDATE_COMP_N_LABEL(1,k.g.n.edge_u);
	UPDATE_COMP_N_LABEL(2,k.g.n.edge_r);
	UPDATE_COMP_N_LABEL(3,k.g.n.edge_l);
#if NEIGHBORHOOD == 8
	UPDATE_COMP_N_LABEL(4,k.g.n.edge_dr);
	UPDATE_COMP_N_LABEL(5,k.g.n.edge_dl);
	UPDATE_COMP_N_LABEL(6,k.g.n.edge_ur);
	UPDATE_COMP_N_LABEL(7,k.g.n.edge_ul);
#endif
	k.g.n.comp_n[thread_id] = comp_n;
}

__global__ void SpreadLabels(KernelWrapper k, int * label, int * alive) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int thread_id = x + y * k.g.width_ex;
	int label_id = x + y * k.g.width;
	int local_idx = (threadIdx.y + 1) * 34 + threadIdx.x + 1;
	
	int comp_n = k.g.n.comp_n[thread_id];

	__shared__ int local_label[356];
	
	if (x < k.g.width && y < k.g.height) {

		local_label[local_idx] = label[label_id];

		threadIdx.x == 31 && x < k.g.width - 1 ? local_label[local_idx + 1]
				= label[label_id + 1] : 0;
		threadIdx.x == 0 && x > 0 ? local_label[local_idx - 1]
				= label[label_id - 1] : 0;
		threadIdx.y == 7 && y < k.g.height - 1 ? local_label[local_idx + 34]
				= label[label_id + k.g.width] : 0;
		threadIdx.y == 0 && y > 0 ? local_label[local_idx - 34]
				= label[label_id - k.g.width] : 0;

#if NEIGHBORHOOD == 8
		threadIdx.x == 0  && threadIdx.y == 0 &&
		x > 0 && y > 0 ? local_label[local_idx - 35] = label[label_id - 1 - k.g.width] : 0;
		threadIdx.x == 31 && threadIdx.y == 0 &&
		x < k.g.width - 1 && y > 0 ? local_label[local_idx - 33] = label[label_id + 1 - k.g.width] : 0;
		threadIdx.x == 0  && threadIdx.y == 7 &&
		x > 0 && y < k.g.height - 1 ? local_label[local_idx + 33] = label[label_id - 1 + k.g.width] : 0;
		threadIdx.x == 31 && threadIdx.y == 7 &&
		x < k.g.width - 1 && y < k.g.height - 1 ? local_label[local_idx + 35] = label[label_id + 1 + k.g.width] : 0;
#endif
	}
	
	__syncthreads();

	int curr_label = local_label[local_idx];
	int orig_label = curr_label;
	
	int repetitions = 4;

	do{
		if (x > 0 && y > 0 && x < k.g.width - 1 && y < k.g.height - 1) {

#ifdef SPREAD_ZEROS
			curr_label = ( (comp_n & (1<<0)) || local_label[local_idx+34]) && curr_label;
			curr_label = ( (comp_n & (1<<1)) || local_label[local_idx-34]) && curr_label;
			curr_label = ( (comp_n & (1<<2)) || local_label[local_idx+1]) && curr_label;
			curr_label = ( (comp_n & (1<<3)) || local_label[local_idx-1]) && curr_label;
	#if NEIGHBORHOOD == 8
			curr_label = ( (comp_n & (1<<4)) || local_label[local_idx+35]) && curr_label;
			curr_label = ( (comp_n & (1<<5)) || local_label[local_idx+33]) && curr_label;
			curr_label = ( (comp_n & (1<<6)) || local_label[local_idx-33]) && curr_label;
			curr_label = ( (comp_n & (1<<7)) || local_label[local_idx-35]) && curr_label;
	#endif
#else
			curr_label = k.g.n.edge_u[thread_id+k.g.width_ex] && local_label[local_idx+34] || curr_label;
			curr_label = k.g.n.edge_d[thread_id-k.g.width_ex] && local_label[local_idx-34] || curr_label;
			curr_label = k.g.n.edge_l[thread_id+1] && local_label[local_idx+1] || curr_label;
			curr_label = k.g.n.edge_r[thread_id-1] && local_label[local_idx-1] || curr_label;
	#if NEIGHBORHOOD == 8
			curr_label = k.g.n.edge_ul[thread_id+k.g.width_ex+1] && local_label[local_idx+35] || curr_label;
			curr_label = k.g.n.edge_ur[thread_id+k.g.width_ex-1] && local_label[local_idx+33] || curr_label;
			curr_label = k.g.n.edge_dl[thread_id-k.g.width_ex+1] && local_label[local_idx-33] || curr_label;
			curr_label = k.g.n.edge_dr[thread_id-k.g.width_ex-1] && local_label[local_idx-35] || curr_label;
	#endif
#endif
			curr_label != orig_label ? label[label_id] = curr_label : 0;
			curr_label != orig_label ? alive[0] = 1 : 0;
			orig_label = curr_label;
		}
		__syncthreads();
	} while(--repetitions);
}

__global__ void Push(KernelWrapper k, int iter, int skip, int * alive) {
	if (!skip || k.active[blockIdx.x + blockIdx.y * k.block_x]) {
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;
		int thread_id = x + y * k.g.width_ex;

		int comp_h = k.g.n.comp_h[thread_id];

		int local_idx = (threadIdx.y + 1) * 34 + threadIdx.x + 1;

		/*__shared__ int local_height[356];

		local_height[local_idx] = k.g.n.height[thread_id];

		threadIdx.x == 31 && x < k.g.width_ex - 1 ? local_height[local_idx + 1]
				= k.g.n.height[thread_id + 1] : 0;
		threadIdx.x == 0 && x > 0 ? local_height[local_idx - 1]
				= k.g.n.height[thread_id - 1] : 0;
		threadIdx.y == 7 && y < k.g.height_ex - 1 ? local_height[local_idx + 34]
		        = k.g.n.height[thread_id + k.g.width_ex] : 0;
		threadIdx.y == 0 && y > 0 ? local_height[local_idx - 34]
				= k.g.n.height[thread_id - k.g.width_ex] : 0;

#if NEIGHBORHOOD == 8
		threadIdx.x == 0 && threadIdx.y == 0 &&
		x > 0 && y > 0 ? local_height[local_idx - 35] = k.g.n.height[thread_id - 1 - k.g.width_ex] : 0;
		threadIdx.x == 31 && threadIdx.y == 0 &&
		x < k.g.width_ex - 1 && y > 0 ? local_height[local_idx - 33] = k.g.n.height[thread_id + 1 - k.g.width_ex] : 0;
		threadIdx.x == 0 && threadIdx.y == 7 &&
		x > 0 && y < k.g.height_ex - 1 ? local_height[local_idx + 33] = k.g.n.height[thread_id - 1 + k.g.width_ex] : 0;
		threadIdx.x == 31 && threadIdx.y == 7 &&
		x < k.g.width_ex - 1 && y < k.g.height_ex - 1 ? local_height[local_idx + 35] = k.g.n.height[thread_id + 1 + k.g.width_ex] : 0;
#endif*/

		__shared__ int local_excess[356];

		local_excess[local_idx] = k.g.n.excess[thread_id];

		threadIdx.x == 31 && x < k.g.width_ex - 1  ? local_excess[local_idx +  1] = 0 : 0;
		threadIdx.x == 0  && x > 0                 ? local_excess[local_idx -  1] = 0 : 0;
		threadIdx.y == 7  && y < k.g.height_ex - 1 ? local_excess[local_idx + 34] = 0 : 0;
		threadIdx.y == 0  && y > 0                 ? local_excess[local_idx - 34] = 0 : 0;

#if NEIGHBORHOOD == 8
		threadIdx.x == 0 && threadIdx.y == 0 && x > 0                && y > 0                 ? local_excess[local_idx -35] = 0 : 0;
		threadIdx.x == 31&& threadIdx.y == 0 && x < k.g.width_ex - 1 && y > 0                 ? local_excess[local_idx -33] = 0 : 0;
		threadIdx.x == 0 && threadIdx.y == 7 && x > 0                && y < k.g.height_ex - 1 ? local_excess[local_idx +33] = 0 : 0;
		threadIdx.x == 31&& threadIdx.y == 7 && x < k.g.width_ex - 1 && y < k.g.height_ex - 1 ? local_excess[local_idx +35] = 0 : 0;
#endif

		__syncthreads();
		
		bool did_something = false;

		int original_excess = local_excess[local_idx];

		//if (k.g.n.status[thread_id]) {
			int excess;
			int cap;
			int flow;
			do {

				DO_PUSH(k.g.n.edge_l,k.g.n.edge_r,1,0,0,0,0);
				DO_PUSH(k.g.n.edge_r,k.g.n.edge_l,0,1,0,0,1);
				DO_PUSH(k.g.n.edge_u,k.g.n.edge_d,0,0,1,0,2);
				DO_PUSH(k.g.n.edge_d,k.g.n.edge_u,0,0,0,1,3);
#if NEIGHBORHOOD == 8
				DO_PUSH(k.g.n.edge_dr,k.g.n.edge_ul,0,1,0,1,4);
				DO_PUSH(k.g.n.edge_dl,k.g.n.edge_ur,1,0,0,1,5);
				DO_PUSH(k.g.n.edge_ur,k.g.n.edge_dl,0,1,1,0,6);
				DO_PUSH(k.g.n.edge_ul,k.g.n.edge_dr,1,0,1,0,7);
#endif

				//excess = k.g.n.excess[thread_id];
			} while (--iter);
			excess = local_excess[local_idx];

			/*if(threadIdx.x > 0 && threadIdx.x < 31 && threadIdx.y > 0 && threadIdx.y < 7)
				excess - original_excess ? k.g.n.excess[thread_id] = excess : 0;
			else*/
				excess - original_excess ? atomicAdd(&k.g.n.excess[thread_id], excess - original_excess) : 0;
				
			threadIdx.x == 0 && x > 0 && local_excess[local_idx - 1] ? atomicAdd(&k.g.n.excess[thread_id - 1], local_excess[local_idx - 1]) : 0;
			threadIdx.y == 0 && y > 0 && local_excess[local_idx - 34] ? atomicAdd(&k.g.n.excess[thread_id - k.g.width_ex], local_excess[local_idx - 34]) : 0;
			threadIdx.x == 31 && x < k.g.width_ex - 1 && local_excess[local_idx + 1] ? atomicAdd(&k.g.n.excess[thread_id + 1], local_excess[local_idx + 1]) : 0;
			threadIdx.y == 7 && y < k.g.height_ex - 1 && local_excess[local_idx + 34] ? atomicAdd(&k.g.n.excess[thread_id + k.g.width_ex], local_excess[local_idx + 34]) : 0;
#if NEIGHBORHOOD == 8
#endif
		//}
		if(!skip && did_something)
			alive[0] = 1;
	}
}

#define ADJUST_HEIGHT(diff,edge) (height > local_height[local_idx + (diff)] && (edge)[thread_id] > 0) ? height = local_height[local_idx + (diff)] : 0

#define UPDATE_COMP_H(i,diff) comp_h |= (1 << (i)) * (local_height[local_idx] == local_height[local_idx+(diff)] + 1)

__global__ void Relabel(KernelWrapper k, int skip) {
	if (!skip || k.active[blockIdx.x + blockIdx.y * k.block_x]) {
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;
		int thread_id = x + y * k.g.width_ex;

		__shared__ int local_height[356];

		int local_idx = (threadIdx.y + 1) * 34 + threadIdx.x + 1;

		local_height[local_idx] = k.g.n.height[thread_id];

		threadIdx.x == 31 && x < k.g.width_ex - 1 ? local_height[local_idx + 1]
				= k.g.n.height[thread_id + 1] : 0;
		threadIdx.x == 0 && x > 0 ? local_height[local_idx - 1]
				= k.g.n.height[thread_id - 1] : 0;
		threadIdx.y == 7 && y < k.g.height_ex - 1 ? local_height[local_idx + 34]
		        = k.g.n.height[thread_id + k.g.width_ex] : 0;
		threadIdx.y == 0 && y > 0 ? local_height[local_idx - 34]
				= k.g.n.height[thread_id - k.g.width_ex] : 0;

		#if NEIGHBORHOOD == 8
		threadIdx.x == 0 && threadIdx.y == 0 &&
		x > 0 && y > 0 ? local_height[local_idx - 35] = k.g.n.height[thread_id - 1 - k.g.width_ex] : 0;
		threadIdx.x == 31 && threadIdx.y == 0 &&
		x < k.g.width_ex - 1 && y > 0 ? local_height[local_idx - 33] = k.g.n.height[thread_id + 1 - k.g.width_ex] : 0;
		threadIdx.x == 0 && threadIdx.y == 7 &&
		x > 0 && y < k.g.height_ex - 1 ? local_height[local_idx + 33] = k.g.n.height[thread_id - 1 + k.g.width_ex] : 0;
		threadIdx.x == 31 && threadIdx.y == 7 &&
		x < k.g.width_ex - 1 && y < k.g.height_ex - 1 ? local_height[local_idx + 35] = k.g.n.height[thread_id + 1 + k.g.width_ex] : 0;
		#endif
		
		__syncthreads();
		
		int excess = k.g.n.excess[thread_id];
		//int status = 0;

		if (excess >= 0 && x > 0 && y > 0 && x < k.g.width - 1 && y < k.g.height - 1) {
			int height = k.g.size;

			ADJUST_HEIGHT( -1,k.g.n.edge_l);
			ADJUST_HEIGHT(  1,k.g.n.edge_r);
			ADJUST_HEIGHT(-34,k.g.n.edge_u);
			ADJUST_HEIGHT( 34,k.g.n.edge_d);
#if NEIGHBORHOOD == 8
			ADJUST_HEIGHT(-35,k.g.n.edge_ul);
			ADJUST_HEIGHT(-33,k.g.n.edge_ur);
			ADJUST_HEIGHT( 33,k.g.n.edge_dl);
			ADJUST_HEIGHT( 35,k.g.n.edge_dr);
#endif
			//height != k.g.size_ex ? printf("Changed from %d to %d\n",k.g.n.height[thread_id],height): 0;
			//status = height != k.g.size;
			k.g.n.height[thread_id] = height + 1;
			//__sync?
			local_height[local_idx] = height + 1;
		}

		__syncthreads();
		int comp_h = 0;
		if (x > 0 && y > 0 && x < k.g.width - 1 && y < k.g.height - 1) {
			UPDATE_COMP_H(0, -1);
			UPDATE_COMP_H(1,  1);
			UPDATE_COMP_H(2,-34);
			UPDATE_COMP_H(3, 34);
	#if NEIGHBORHOOD == 8
			UPDATE_COMP_H(7,-35);
			UPDATE_COMP_H(6,-33);
			UPDATE_COMP_H(5, 33);
			UPDATE_COMP_H(4, 35);
	#endif
		}
		k.g.n.comp_h[thread_id] = comp_h;

		k.g.n.status[thread_id] = excess > 0 && local_height[local_idx] != k.g.size + 1;
		//k.g.n.status[thread_id] = status;
	}
}

__global__ void UpdateActivity(int * status, int * active, int block_x, int width_ex) {
	int block_id = blockIdx.x + blockIdx.y * block_x;
	active[block_id] = 0;

	__syncthreads();

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int thread_id = x + y * width_ex;

	status[thread_id] ? active[block_id] = 1 : 0;
}

#define UPDATE_COMP_N(i,edge) comp_n |= (1<<(i)) * (edge[thread_id] > 0)

__global__ void InitGlobalRelabel(KernelWrapper k) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int thread_id = x + y * k.g.width_ex;

	int no_sink = 0;
	if (x > 0 && y > 0 && x < k.g.width - 1 && y < k.g.height - 1) {
		no_sink = k.g.n.excess[thread_id] >= 0;
		k.g.n.height[thread_id] = no_sink * k.g.size + 1;
	}

	int comp_n = 0;
	UPDATE_COMP_N(0,k.g.n.edge_l);
	UPDATE_COMP_N(1,k.g.n.edge_r);
	UPDATE_COMP_N(2,k.g.n.edge_u);
	UPDATE_COMP_N(3,k.g.n.edge_d);
#if NEIGHBORHOOD == 8
	UPDATE_COMP_N(4,k.g.n.edge_ul);
	UPDATE_COMP_N(5,k.g.n.edge_ur);
	UPDATE_COMP_N(6,k.g.n.edge_dl);
	UPDATE_COMP_N(7,k.g.n.edge_dr);
#endif
	/* different because of >= instead of > */
	/* and accounting for activity instead of only no-sinkness */
	comp_n |= (1<<8) * no_sink;
	k.g.n.comp_n[thread_id] = comp_n;
}

#define COMP_ADJUST_HEIGHT(diff,i) (height > local_height[local_idx + (diff)] && ((1<<(i)) & comp_n)) ? height = local_height[local_idx + (diff)] : 0

__global__ void GlobalRelabel(KernelWrapper k, int * alive) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int thread_id = x + y * k.g.width_ex;

	int comp_n = k.g.n.comp_n[thread_id];
	if ( __syncthreads_or(comp_n & (1<<8)) ) {

		__shared__ int local_height[356];

		int local_idx = (threadIdx.y + 1) * 34 + threadIdx.x + 1;

		local_height[local_idx] = k.g.n.height[thread_id];

		/*int outer_repetitions = 1;

		do {*/
			threadIdx.x == 31 && x < k.g.width_ex - 1 ?
					local_height[local_idx + 1] = k.g.n.height[thread_id + 1] : 0;
			threadIdx.x == 0 && x > 0 ?
					local_height[local_idx - 1] = k.g.n.height[thread_id - 1] : 0;
			threadIdx.y == 7 && y < k.g.height_ex - 1 ?
					local_height[local_idx + 34] =
							k.g.n.height[thread_id + k.g.width_ex] :
					0;
			threadIdx.y == 0 && y > 0 ?
					local_height[local_idx - 34] =
							k.g.n.height[thread_id - k.g.width_ex] :
					0;

		#if NEIGHBORHOOD == 8
			threadIdx.x == 0 && threadIdx.y == 0 &&
			x > 0 && y > 0 ? local_height[local_idx - 35] = k.g.n.height[thread_id - 1 - k.g.width_ex] : 0;
			threadIdx.x == 31 && threadIdx.y == 0 &&
			x < k.g.width_ex - 1 && y > 0 ? local_height[local_idx - 33] = k.g.n.height[thread_id + 1 - k.g.width_ex] : 0;
			threadIdx.x == 0 && threadIdx.y == 7 &&
			x > 0 && y < k.g.height_ex - 1 ? local_height[local_idx + 33] = k.g.n.height[thread_id - 1 + k.g.width_ex] : 0;
			threadIdx.x == 31 && threadIdx.y == 7 &&
			x < k.g.width_ex - 1 && y < k.g.height_ex - 1 ? local_height[local_idx + 35] = k.g.n.height[thread_id + 1 + k.g.width_ex] : 0;
		#endif

			__syncthreads();

				bool changed = false;

				int repetitions = 10;
				do {
					int height = local_height[local_idx] - 1;
					if (((1<<8) & comp_n) && x > 0 && y > 0	&& x < k.g.width - 1 && y < k.g.height - 1) {
						height = k.g.size;

						COMP_ADJUST_HEIGHT( -1, 0);
						COMP_ADJUST_HEIGHT(  1, 1);
						COMP_ADJUST_HEIGHT(-34, 2);
						COMP_ADJUST_HEIGHT( 34, 3);
			#if NEIGHBORHOOD == 8
						COMP_ADJUST_HEIGHT(-35, 4);
						COMP_ADJUST_HEIGHT(-33, 5);
						COMP_ADJUST_HEIGHT( 33, 6);
						COMP_ADJUST_HEIGHT( 35, 7);
			#endif
					}
					__syncthreads();
					changed |= (local_height[local_idx] != height + 1);
					local_height[local_idx] = height + 1;
					__syncthreads();
				} while (--repetitions);
				//height != k.g.size_ex ? printf("Changed from %d to %d\n",k.g.n.height[thread_id],height): 0;
				changed ? k.g.n.comp_n[thread_id] = comp_n ^ (1<<8) : 0;
				changed ? k.g.n.height[thread_id] = local_height[local_idx] : 0;
				//k.g.n.height[thread_id] = height + 1;

				//__syncthreads();

				changed ? alive[0] = 1 : 0;
		//} while(--outer_repetitions);
	}
}

#endif /* GRAPHCUTKERNELS_CU_ */
